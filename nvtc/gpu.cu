#include "hip/hip_runtime.h"
#include "gpu.h"
#include "gpu-thrust.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <thread>

#include "TrCountingGraph.h"
using namespace std;

#define NUM_THREADS 64
#define NUM_BLOCKS_GENERIC 112
#define NUM_BLOCKS_PER_MP 8

void CudaAssert(hipError_t status, const char* code, const char* file, int l) {
  if (status == hipSuccess) return;
  cerr << "Cuda error: " << code << ", file " << file << ", line " << l << endl;
  exit(1);
}

#define CUCHECK(x) CudaAssert(x, #x, __FILE__, __LINE__)

__global__ void CalculateTrianglesSplit_v2(TrCountingGraphChunk* chunk, int deviceCount = 1, int deviceIdx = 0) {
    int from =
    gridDim.x * blockDim.x * deviceIdx +
    blockDim.x * blockIdx.x +
    threadIdx.x;
  int step = deviceCount * gridDim.x * blockDim.x;
  
  uint64_t count = 0;
  for (int k = from; k < chunk->length; k += step) {
    int i =  chunk->dev_neighbor_start_i[k]; 
    int j =  chunk->dev_neighbor_i[k]; 
    if (j == 2147483647 || chunk->dev_offset[j] < chunk->dev_split_offset[chunk->chunkid_j] || chunk->dev_offset[j] >= chunk->dev_split_offset[chunk->chunkid_j+1]) 
        continue;
    int64_t j_it = chunk->dev_offset[j]-chunk->dev_split_offset[chunk->chunkid_j];
    int64_t i_it = chunk->dev_offset[i]-chunk->dev_split_offset[chunk->chunkid_i];
    int64_t j_it_end = j_it+chunk->dev_degree[j]-1;
    int64_t i_it_end = i_it+chunk->dev_degree[i]-1;

    int a = chunk->dev_neighbor_i[i_it], b = chunk->dev_neighbor_j[j_it]; 
    while(j_it <= j_it_end && i_it <= i_it_end) {
      int d = a - b;
      if(d == 0) {
        count++;
      }
      if(d <= 0)
        a = chunk->dev_neighbor_i[++i_it]; 
      if(d >= 0)
        b = chunk->dev_neighbor_j[++j_it];
    }
  }
  chunk->dev_results[blockDim.x * blockIdx.x + threadIdx.x] = count;
}

__global__ void CalculateTriangles_v2(TrCountingGraphChunk* chunk, int n, uint64_t* results, int deviceCount = 1,
    int deviceIdx = 0) {
    int from =
    gridDim.x * blockDim.x * deviceIdx +
    blockDim.x * blockIdx.x +
    threadIdx.x;
  int step = deviceCount * gridDim.x * blockDim.x;
  
  uint64_t count = 0;
  for (int k = from; k < n; k += step) {
    int i =  chunk->dev_neighbor_start[k]; 
    int j =  chunk->dev_neighbor[k]; 
    if (j==2147483647) continue;
    int64_t j_it = chunk->dev_offset[j];
    int64_t i_it = chunk->dev_offset[i];
    int64_t j_it_end = j_it+chunk->dev_degree[j]-1;
    int64_t i_it_end = i_it+chunk->dev_degree[i]-1;

    int a = chunk->dev_neighbor[i_it], b = chunk->dev_neighbor[j_it]; 
    while(j_it <= j_it_end && i_it <= i_it_end){
      int d = a-b;
      if ( d == 0 ){
        count++;
      }
      if (d <= 0)
        a = chunk->dev_neighbor[++i_it]; 
      if (d >= 0)
        b = chunk->dev_neighbor[++j_it];
    }
  }
  results[blockDim.x * blockIdx.x + threadIdx.x] = count;
}

int NumberOfMPs() {
  int dev, val;
  CUCHECK(hipGetDevice(&dev));
  CUCHECK(hipDeviceGetAttribute(&val, hipDeviceAttributeMultiprocessorCount, dev));
  return val;
}

size_t GlobalMemory() {
  int dev;
  hipDeviceProp_t prop;
  CUCHECK(hipGetDevice(&dev));
  CUCHECK(hipGetDeviceProperties(&prop, dev));
  return prop.totalGlobalMem;
}

uint64_t GpuForward_v2(const TrCountingGraph& TrCountingGraph) {
    const int NUM_BLOCKS = NUM_BLOCKS_PER_MP * NumberOfMPs();
    TrCountingGraphChunk chunk(TrCountingGraph, 1, TrCountingGraph.edge_num);

    uint64_t* dev_results;
    CUCHECK(hipMalloc(&dev_results,
          NUM_BLOCKS * NUM_THREADS * sizeof(uint64_t)));
    
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(CalculateTriangles_v2), hipFuncCachePreferL1);
    CalculateTriangles_v2<<<NUM_BLOCKS, NUM_THREADS>>>(chunk.dev_this, 
        TrCountingGraph.offset[TrCountingGraph.nodeid_max+1], dev_results);
    CUCHECK(hipDeviceSynchronize());
    uint64_t result = SumResults(NUM_BLOCKS * NUM_THREADS, dev_results);
    return result;
}

int GetDevNum(){
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  return deviceCount;
}

int GetSplitNum(int num_nodes, uint64_t num_edges) {
  int DevNum = GetDevNum();
  int split_num = 0;
  int split_num_new = 0;
  for (int i = 0; i < DevNum; i++){
    CUCHECK(hipSetDevice(i));
    uint64_t mem = (uint64_t)GlobalMemory();  // in Byte
    mem -= (uint64_t)num_nodes * 16;  // uint64_t
    split_num_new = (int)(1 + 12 * (num_edges) / mem);
    split_num = split_num > split_num_new ? split_num : split_num_new;
  }
  return split_num;
}
void calculation_thread(TrCountingGraph* TrCountingGraph, int split_num, int64_t cpu_offset,int gpu_offset_start, int gpu_offset_end, int rank, int step, int64_t* result_temp){
  CUCHECK(hipSetDevice(rank));
  const int NUM_BLOCKS = NUM_BLOCKS_PER_MP * NumberOfMPs();
  TrCountingGraphChunk chunk(*TrCountingGraph, split_num, cpu_offset);
  int64_t result=0;
  for(int ij = gpu_offset_start+rank; ij < gpu_offset_end; ij=ij+step) {
      int i, j;
      get_i_j(split_num, ij, &i, &j);
      if(chunk.split_offset[i] >= cpu_offset)
          break;
      chunk.initChunk(i, j);
      
      CalculateTrianglesSplit_v2<<<NUM_BLOCKS, NUM_THREADS>>>(chunk.dev_this);
      CUCHECK(hipDeviceSynchronize());
      result = result + SumResults(NUM_BLOCKS * NUM_THREADS, chunk.dev_results);
  }
  *result_temp = result;
}

uint64_t GpuForwardSplit_v2(TrCountingGraph& TrCountingGraph, 
    int split_num, int64_t cpu_offset,
    int gpu_offset_start, int gpu_offset_end) {
    
    int DevNum = GetDevNum();
    int64_t result = 0;
    int64_t* result_temp = new int64_t[DevNum];
    thread** thread_list = new thread*[DevNum];
    for(int i=0;i<DevNum;i++){
      thread_list[i] = new thread(calculation_thread,&TrCountingGraph,split_num,cpu_offset,gpu_offset_start,gpu_offset_end,i,DevNum,result_temp+i);
    }
    for(int i=0;i<DevNum;i++){
      thread_list[i]->join();
      result += result_temp[i];
    }
    return result;
}

TrCountingGraphChunk::TrCountingGraphChunk(const TrCountingGraph& g, int split_num, int64_t cpu_task){
  chunk_length_max = get_split_v2(g.offset, g.nodeid_max, split_num, split_offset);
  // device memory initialization
  const int NUM_BLOCKS = NUM_BLOCKS_PER_MP * NumberOfMPs();
  cpu_offset = cpu_task;
  CUCHECK(hipMalloc(&dev_results, NUM_BLOCKS * NUM_THREADS * sizeof(uint64_t)));
  CUCHECK(hipMalloc(&dev_offset, (g.nodeid_max + 2) * sizeof(int64_t)));
  CUCHECK(hipMemcpy(
      dev_offset, g.offset, (g.nodeid_max + 2) * sizeof(int64_t), hipMemcpyHostToDevice));
  CUCHECK(hipMalloc(&dev_degree, (g.nodeid_max + 1) * sizeof(int)));
  CUCHECK(hipMemcpy(
      dev_degree, g.degree, (g.nodeid_max + 1) * sizeof(int), hipMemcpyHostToDevice));
  CUCHECK(hipMalloc(&dev_split_offset, (split_num + 1) * sizeof(int64_t)));
  CUCHECK(hipMemcpy(
      dev_split_offset, split_offset, (split_num + 1) * sizeof(int64_t), hipMemcpyHostToDevice));
  if(split_num==1){
    CUCHECK(hipMalloc(&dev_neighbor, (g.edge_num) * sizeof(int)));
    CUCHECK(hipMemcpy(
       dev_neighbor, g.neighboor, (g.edge_num) * sizeof(int), hipMemcpyHostToDevice));
    CUCHECK(hipMalloc(&dev_neighbor_start, (g.edge_num) * sizeof(int)));
    CUCHECK(hipMemcpy(
       dev_neighbor_start, g.neighboor_start, (g.edge_num) * sizeof(int), hipMemcpyHostToDevice));
  }
  else{
    CUCHECK(hipMalloc(&dev_neighbor_i, chunk_length_max * sizeof(int)));
    CUCHECK(hipMalloc(&dev_neighbor_start_i, chunk_length_max * sizeof(int)));
    CUCHECK(hipMalloc(&dev_neighbor_j, chunk_length_max * sizeof(int)));
  }
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(CalculateTrianglesSplit_v2), hipFuncCachePreferL1);

  Graph = &g;
  CUCHECK(hipMalloc(&dev_this, sizeof(TrCountingGraphChunk)));
  CUCHECK(hipMemcpy(dev_this,this,sizeof(TrCountingGraphChunk),hipMemcpyHostToDevice));
}
void TrCountingGraphChunk::initChunk(int i, int j){
  chunkid_i = i;
  chunkid_j = j;
  length = cpu_offset>split_offset[i+1]?split_offset[i+1]-split_offset[i]:cpu_offset-split_offset[i];
  CUCHECK(hipMemcpy(
    dev_neighbor_i, Graph->neighboor+split_offset[i], (split_offset[i+1]-split_offset[i])*sizeof(int), hipMemcpyHostToDevice));
  CUCHECK(hipMemcpy(
    dev_neighbor_start_i, Graph->neighboor_start+split_offset[i], (split_offset[i+1]-split_offset[i])*sizeof(int), hipMemcpyHostToDevice));
  CUCHECK(hipMemcpy(
    dev_neighbor_j, Graph->neighboor+split_offset[j], (split_offset[j+1]-split_offset[j])*sizeof(int), hipMemcpyHostToDevice));
  CUCHECK(hipMemcpy(dev_this,this,sizeof(TrCountingGraphChunk),hipMemcpyHostToDevice));
}