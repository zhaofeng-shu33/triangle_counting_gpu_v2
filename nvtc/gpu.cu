#include "hip/hip_runtime.h"
#include "gpu.h"

#include "gpu-thrust.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "TrCountingGraph.h"
using namespace std;

#define NUM_THREADS 64
#define NUM_BLOCKS_GENERIC 112
#define NUM_BLOCKS_PER_MP 8

template<bool ZIPPED>
__global__ void CalculateNodePointers(int n, int m, int* edges, int* nodes) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i <= m; i += step) {
    int prev = i > 0 ? edges[ZIPPED ? (2 * (i - 1) + 1) : (m + i - 1)] : -1;
    int next = i < m ? edges[ZIPPED ? (2 * i + 1) : (m + i)] : n;
    for (int j = prev + 1; j <= next; ++j)
      nodes[j] = i;
  }
}

__global__ void CalculateFlags(int m, int* edges, int* nodes, bool* flags) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < m; i += step) {
    int a = edges[2 * i];
    int b = edges[2 * i + 1];
    int deg_a = nodes[a + 1] - nodes[a];
    int deg_b = nodes[b + 1] - nodes[b];
    flags[i] = (deg_a < deg_b) || (deg_a == deg_b && a < b);
  }
}

__global__ void UnzipEdges(int m, int* edges, int* unzipped_edges) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < m; i += step) {
    unzipped_edges[i] = edges[2 * i];
    unzipped_edges[m + i] = edges[2 * i + 1];
  }
}
__global__ void CalculateTrianglesSplit_v2(int n,const int64_t* __restrict__ dev_offset,
    const int* __restrict__ dev_length,const int* __restrict__ dev_neighbor_i,
    const int* __restrict__ dev_neighbor_start_i, const int* __restrict__ dev_neighbor_j,
    const int64_t* __restrict__ dev_split_offset,
    int chunkid_i, int chunkid_j, uint64_t* results, int deviceCount = 1, int deviceIdx = 0) {
    int from =
    gridDim.x * blockDim.x * deviceIdx +
    blockDim.x * blockIdx.x +
    threadIdx.x;
  int step = deviceCount * gridDim.x * blockDim.x;
  
  uint64_t count = 0;
  for (int k = from; k < n; k += step) {
    int i =  dev_neighbor_start_i[k]; 
    int j =  dev_neighbor_i[k]; 
    if (j == 2147483647 || dev_offset[j] < dev_split_offset[chunkid_j] || dev_offset[j] >= dev_split_offset[chunkid_j+1]) 
        continue;
    int64_t j_it = dev_offset[j]-dev_split_offset[chunkid_j];
    int64_t i_it = dev_offset[i]-dev_split_offset[chunkid_i];
    int64_t j_it_end = j_it+dev_length[j]-1;
    int64_t i_it_end = i_it+dev_length[i]-1;

    int a = dev_neighbor_i[i_it], b = dev_neighbor_j[j_it]; 
    while(j_it <= j_it_end && i_it <= i_it_end) {
      int d = a - b;
      if(d == 0) {
        count++;
      }
      if(d <= 0)
        a = dev_neighbor_i[++i_it]; 
      if(d >= 0)
        b = dev_neighbor_j[++j_it];
    }
  }
  results[blockDim.x * blockIdx.x + threadIdx.x] = count;
}

__global__ void CalculateTriangles_v2(int n,const int* __restrict__ dev_neighbor,
    const int64_t* __restrict__ dev_offset,const int* __restrict__ dev_length,
    const int* __restrict__ dev_neighbor_start, uint64_t* results,int deviceCount = 1,
    int deviceIdx = 0) {
    int from =
    gridDim.x * blockDim.x * deviceIdx +
    blockDim.x * blockIdx.x +
    threadIdx.x;
  int step = deviceCount * gridDim.x * blockDim.x;
  
  uint64_t count = 0;
  for (int k = from; k < n; k += step) {
    int i =  dev_neighbor_start[k]; 
    int j =  dev_neighbor[k]; 
    if (j==2147483647) continue;
    int64_t j_it = dev_offset[j];
    int64_t i_it = dev_offset[i];
    int64_t j_it_end = j_it+dev_length[j]-1;
    int64_t i_it_end = i_it+dev_length[i]-1;

    int a = dev_neighbor[i_it], b = dev_neighbor[j_it]; 
    while(j_it <= j_it_end && i_it <= i_it_end){
      int d = a-b;
      if ( d == 0 ){
        count++;
      }
      if (d <= 0)
        a = dev_neighbor[++i_it]; 
      if (d >= 0)
        b = dev_neighbor[++j_it];
    }
  }
  results[blockDim.x * blockIdx.x + threadIdx.x] = count;
}

__global__ void CalculateTriangles(
    int m, const int* __restrict__ edges, const int* __restrict__ nodes,
    uint64_t* results, int deviceCount = 1, int deviceIdx = 0) {
  int from =
    gridDim.x * blockDim.x * deviceIdx +
    blockDim.x * blockIdx.x +
    threadIdx.x;
  int step = deviceCount * gridDim.x * blockDim.x;
  uint64_t count = 0;

  for (int i = from; i < m; i += step) {
    int u = edges[i], v = edges[m + i];

    int u_it = nodes[u], u_end = nodes[u + 1];
    int v_it = nodes[v], v_end = nodes[v + 1];

    int a = edges[u_it], b = edges[v_it];
    while (u_it < u_end && v_it < v_end) {
      int d = a - b;
      if (d <= 0)
        a = edges[++u_it];
      if (d >= 0)
        b = edges[++v_it];
      if (d == 0)
        ++count;
    }
  }

  results[blockDim.x * blockIdx.x + threadIdx.x] = count;
}

void CudaAssert(hipError_t status, const char* code, const char* file, int l) {
  if (status == hipSuccess) return;
  cerr << "Cuda error: " << code << ", file " << file << ", line " << l << endl;
  exit(1);
}

#define CUCHECK(x) CudaAssert(x, #x, __FILE__, __LINE__)

int NumberOfMPs() {
  int dev, val;
  CUCHECK(hipGetDevice(&dev));
  CUCHECK(hipDeviceGetAttribute(&val, hipDeviceAttributeMultiprocessorCount, dev));
  return val;
}

size_t GlobalMemory() {
  int dev;
  hipDeviceProp_t prop;
  CUCHECK(hipGetDevice(&dev));
  CUCHECK(hipGetDeviceProperties(&prop, dev));
  return prop.totalGlobalMem;
}



uint64_t MultiGPUCalculateTriangles(
    int n, int m, int* dev_edges, int* dev_nodes, int device_count) {
  vector<int*> multi_dev_edges(device_count);
  vector<int*> multi_dev_nodes(device_count);

  multi_dev_edges[0] = dev_edges;
  multi_dev_nodes[0] = dev_nodes;

  for (int i = 1; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipMalloc(&multi_dev_edges[i], m * 2 * sizeof(int)));
    CUCHECK(hipMalloc(&multi_dev_nodes[i], (n + 1) * sizeof(int)));
    int dst = i, src = (i + 1) >> 2;
    CUCHECK(hipMemcpyPeer(
          multi_dev_edges[dst], dst, multi_dev_edges[src], src,
          m * 2 * sizeof(int)));
    CUCHECK(hipMemcpyPeer(
          multi_dev_nodes[dst], dst, multi_dev_nodes[src], src,
          (n + 1) * sizeof(int)));
  }

  vector<int> NUM_BLOCKS(device_count);
  vector<uint64_t*> multi_dev_results(device_count);

  for (int i = 0; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    NUM_BLOCKS[i] = NUM_BLOCKS_PER_MP * NumberOfMPs();
    CUCHECK(hipMalloc(
          &multi_dev_results[i],
          NUM_BLOCKS[i] * NUM_THREADS * sizeof(uint64_t)));
  }

  for (int i = 0; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(CalculateTriangles), hipFuncCachePreferL1));
    CalculateTriangles<<<NUM_BLOCKS[i], NUM_THREADS>>>(
        m, multi_dev_edges[i], multi_dev_nodes[i], multi_dev_results[i],
        device_count, i);
  }

  uint64_t result = 0;

  for (int i = 0; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipDeviceSynchronize());
    result += SumResults(NUM_BLOCKS[i] * NUM_THREADS, multi_dev_results[i]);
  }

  for (int i = 1; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipFree(multi_dev_edges[i]));
    CUCHECK(hipFree(multi_dev_nodes[i]));
  }

  for (int i = 0; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipFree(multi_dev_results[i]));
  }

  hipSetDevice(0);
  return result;
}

uint64_t GpuForward(int* edges, int num_nodes, uint64_t num_edges) {
  return MultiGpuForward(edges, 1, num_nodes, num_edges);
}

uint64_t GpuForward_v2(const TrCountingGraph& TrCountingGraph){
    int64_t* dev_offset;
    int* dev_neighbor;
    int* dev_length;
    int* dev_neighbor_start;
    CUCHECK(hipMalloc(&dev_offset, (TrCountingGraph.nodeid_max + 2) * sizeof(int64_t)));
    CUCHECK(hipMemcpyAsync(
       dev_offset, TrCountingGraph.offset, (TrCountingGraph.nodeid_max + 2) * sizeof(int64_t), hipMemcpyHostToDevice));
    CUCHECK(hipDeviceSynchronize());
    CUCHECK(hipMalloc(&dev_length, (TrCountingGraph.nodeid_max + 1) * sizeof(int)));
    CUCHECK(hipMemcpyAsync(
      dev_length, TrCountingGraph.degree, (TrCountingGraph.nodeid_max + 1) * sizeof(int), hipMemcpyHostToDevice));
    CUCHECK(hipDeviceSynchronize());
    CUCHECK(hipMalloc(&dev_neighbor, ( TrCountingGraph.edge_num) * sizeof(int)));
    CUCHECK(hipMemcpyAsync(
       dev_neighbor, TrCountingGraph.neighboor, ( TrCountingGraph.edge_num) * sizeof(int), hipMemcpyHostToDevice));
    CUCHECK(hipDeviceSynchronize());
    CUCHECK(hipMalloc(&dev_neighbor_start, ( TrCountingGraph.edge_num) * sizeof(int)));
    CUCHECK(hipMemcpyAsync(
       dev_neighbor_start, TrCountingGraph.neighboor_start, ( TrCountingGraph.edge_num) * sizeof(int), hipMemcpyHostToDevice));
    CUCHECK(hipDeviceSynchronize());
    const int NUM_BLOCKS = NUM_BLOCKS_PER_MP * NumberOfMPs();	
    uint64_t* dev_results;
    CUCHECK(hipMalloc(&dev_results,
          NUM_BLOCKS * NUM_THREADS * sizeof(uint64_t)));
    
    hipSetDevice(0);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(CalculateTriangles_v2), hipFuncCachePreferL1);
    CalculateTriangles_v2<<<NUM_BLOCKS, NUM_THREADS>>>(
        TrCountingGraph.offset[TrCountingGraph.nodeid_max+1], dev_neighbor, dev_offset, dev_length, dev_neighbor_start, dev_results);
    CUCHECK(hipDeviceSynchronize());
    uint64_t result = SumResults(NUM_BLOCKS * NUM_THREADS, dev_results);
    return result;
}

int GetSplitNum(int num_nodes, uint64_t num_edges) {
  uint64_t mem = (uint64_t)GlobalMemory();  // in Byte
  mem -= (uint64_t)num_nodes * 16;  // uint64_t
  return (1 + 12 * (num_edges) / mem);
}

void InitializeGPUMemory() {
    
}

uint64_t GpuForwardSplit_v2(const TrCountingGraph& TrCountingGraph, int split_num, int64_t cpu_offset,
    int gpu_offset_start, int gpu_offset_end) {
    CUCHECK(hipSetDevice(0));
    const int NUM_BLOCKS = NUM_BLOCKS_PER_MP * NumberOfMPs();
  
    // Calculate chunk size
    int64_t* split_offset;
    int64_t chunk_length_max = get_split_v2(TrCountingGraph.offset, TrCountingGraph.nodeid_max, split_num, split_offset);

    // device memory initialization
    int64_t* dev_offset;
    int* dev_length;
    int* dev_neighbor_i;
    int* dev_neighbor_start_i;
    int* dev_neighbor_j;
    CUCHECK(hipMalloc(&dev_offset, (TrCountingGraph.nodeid_max + 2) * sizeof(int64_t)));
    CUCHECK(hipMemcpyAsync(
      dev_offset, TrCountingGraph.offset, (TrCountingGraph.nodeid_max + 2) * sizeof(int64_t), hipMemcpyHostToDevice));
    CUCHECK(hipDeviceSynchronize());
  CUCHECK(hipMalloc(&dev_length, (TrCountingGraph.nodeid_max + 1) * sizeof(int)));
  CUCHECK(hipMemcpyAsync(
    dev_length, TrCountingGraph.degree, (TrCountingGraph.nodeid_max + 1) * sizeof(int), hipMemcpyHostToDevice));
  CUCHECK(hipDeviceSynchronize());
  CUCHECK(hipMalloc(&dev_neighbor_i, chunk_length_max * sizeof(int)));
  CUCHECK(hipMalloc(&dev_neighbor_start_i, chunk_length_max * sizeof(int)));
  CUCHECK(hipMalloc(&dev_neighbor_j, chunk_length_max * sizeof(int)));
  uint64_t* dev_results;
  CUCHECK(hipMalloc(&dev_results, NUM_BLOCKS * NUM_THREADS * sizeof(uint64_t)));
  int64_t* dev_split_offset;
  CUCHECK(hipMalloc(&dev_split_offset, (split_num + 1) * sizeof(int64_t)));
  CUCHECK(hipMemcpyAsync(
    dev_split_offset, split_offset, (split_num + 1) * sizeof(int64_t), hipMemcpyHostToDevice));
  CUCHECK(hipDeviceSynchronize());

  hipSetDevice(0);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(CalculateTrianglesSplit_v2), hipFuncCachePreferL1);
  int64_t result=0;
  for(int ij = gpu_offset_start; ij < gpu_offset_end; ij++) {
      int i, j;
      get_i_j(split_num, ij, &i, &j);
      if(split_offset[i] >= cpu_offset)
          break;
      CUCHECK(hipMemcpyAsync(
         dev_neighbor_i, TrCountingGraph.neighboor+split_offset[i], (split_offset[i+1]-split_offset[i])*sizeof(int), hipMemcpyHostToDevice));
      CUCHECK(hipMemcpyAsync(
        dev_neighbor_start_i, TrCountingGraph.neighboor_start+split_offset[i], (split_offset[i+1]-split_offset[i])*sizeof(int), hipMemcpyHostToDevice));
      CUCHECK(hipMemcpyAsync(
        dev_neighbor_j, TrCountingGraph.neighboor+split_offset[j], (split_offset[j+1]-split_offset[j])*sizeof(int), hipMemcpyHostToDevice));
      CUCHECK(hipDeviceSynchronize());

      CalculateTrianglesSplit_v2<<<NUM_BLOCKS, NUM_THREADS>>>(
        cpu_offset>split_offset[i+1]?split_offset[i+1]-split_offset[i]:cpu_offset-split_offset[i],
        dev_offset, dev_length, dev_neighbor_i, dev_neighbor_start_i, dev_neighbor_j, dev_split_offset, i, j, dev_results);
      CUCHECK(hipDeviceSynchronize());
      result = result + SumResults(NUM_BLOCKS * NUM_THREADS, dev_results);
  }
  return result;
}

uint64_t MultiGpuForward(int* edges, int device_count, int num_nodes, uint64_t num_edges) {
  CUCHECK(hipSetDevice(0));
  const int NUM_BLOCKS = NUM_BLOCKS_PER_MP * NumberOfMPs();

  uint64_t m = num_edges;
  int n = num_nodes;

  int* dev_edges;
  int* dev_nodes;

  
  int* dev_temp;
  CUCHECK(hipMalloc(&dev_temp, m * 2 * sizeof(int)));
  CUCHECK(hipMemcpyAsync(
      dev_temp, edges, m * 2 * sizeof(int), hipMemcpyHostToDevice));
  CUCHECK(hipDeviceSynchronize());
  // Memcpy edges from host to device
  SortEdges(m, dev_temp);
  CUCHECK(hipDeviceSynchronize());
  // Sort edges

  CUCHECK(hipMalloc(&dev_edges, m * 2 * sizeof(int)));
  UnzipEdges<<<NUM_BLOCKS, NUM_THREADS>>>(m, dev_temp, dev_edges);
  CUCHECK(hipFree(dev_temp));
  CUCHECK(hipDeviceSynchronize());
  // Unzip edges


  CUCHECK(hipMalloc(&dev_nodes, (n + 1) * sizeof(int)));
  CalculateNodePointers<false><<<NUM_BLOCKS, NUM_THREADS>>>(
      n, m, dev_edges, dev_nodes);
  CUCHECK(hipDeviceSynchronize());
  // Calculate nodes array for one-way unzipped edges
  uint64_t result = 0;

  if (device_count == 1) {
    uint64_t* dev_results;
    CUCHECK(hipMalloc(&dev_results,
          NUM_BLOCKS * NUM_THREADS * sizeof(uint64_t)));
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(CalculateTriangles), hipFuncCachePreferL1);
    hipProfilerStart();
    CalculateTriangles<<<NUM_BLOCKS, NUM_THREADS>>>(
        m, dev_edges, dev_nodes, dev_results);
    CUCHECK(hipDeviceSynchronize());
    hipProfilerStop();
    // Reduce
    result = SumResults(NUM_BLOCKS * NUM_THREADS, dev_results);
    CUCHECK(hipFree(dev_results));
  } else {
    result = MultiGPUCalculateTriangles(
        n, m, dev_edges, dev_nodes, device_count);
  }

  CUCHECK(hipFree(dev_edges));
  CUCHECK(hipFree(dev_nodes));
  return result;
}

void PreInitGpuContext(int device) {
  CUCHECK(hipSetDevice(device));
  CUCHECK(hipFree(NULL));
}
