#include "hip/hip_runtime.h"
#include "gpu.h"
#include "gpu-thrust.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "TrCountingGraph.h"
using namespace std;

#define NUM_THREADS 64
#define NUM_BLOCKS_GENERIC 112
#define NUM_BLOCKS_PER_MP 8

template<bool ZIPPED>
__global__ void CalculateNodePointers(int n, int m, int* edges, int* nodes) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i <= m; i += step) {
    int prev = i > 0 ? edges[ZIPPED ? (2 * (i - 1) + 1) : (m + i - 1)] : -1;
    int next = i < m ? edges[ZIPPED ? (2 * i + 1) : (m + i)] : n;
    for (int j = prev + 1; j <= next; ++j)
      nodes[j] = i;
  }
}

__global__ void CalculateFlags(int m, int* edges, int* nodes, bool* flags) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < m; i += step) {
    int a = edges[2 * i];
    int b = edges[2 * i + 1];
    int deg_a = nodes[a + 1] - nodes[a];
    int deg_b = nodes[b + 1] - nodes[b];
    flags[i] = (deg_a < deg_b) || (deg_a == deg_b && a < b);
  }
}

__global__ void UnzipEdges(int m, int* edges, int* unzipped_edges) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < m; i += step) {
    unzipped_edges[i] = edges[2 * i];
    unzipped_edges[m + i] = edges[2 * i + 1];
  }
}
__global__ void CalculateTrianglesSplit_v2(TrCountingGraphChunk* chunk, int n, uint64_t* results, int deviceCount = 1, int deviceIdx = 0) {
    int from =
    gridDim.x * blockDim.x * deviceIdx +
    blockDim.x * blockIdx.x +
    threadIdx.x;
  int step = deviceCount * gridDim.x * blockDim.x;
  
  uint64_t count = 0;
  for (int k = from; k < n; k += step) {
    int i =  chunk->dev_neighbor_start_i[k]; 
    int j =  chunk->dev_neighbor_i[k]; 
    if (j == 2147483647 || chunk->dev_offset[j] < chunk->dev_split_offset[chunk->chunkid_j] || chunk->dev_offset[j] >= chunk->dev_split_offset[chunk->chunkid_j+1]) 
        continue;
    int64_t j_it = chunk->dev_offset[j]-chunk->dev_split_offset[chunk->chunkid_j];
    int64_t i_it = chunk->dev_offset[i]-chunk->dev_split_offset[chunk->chunkid_i];
    int64_t j_it_end = j_it+chunk->dev_degree[j]-1;
    int64_t i_it_end = i_it+chunk->dev_degree[i]-1;

    int a = chunk->dev_neighbor_i[i_it], b = chunk->dev_neighbor_j[j_it]; 
    while(j_it <= j_it_end && i_it <= i_it_end) {
      int d = a - b;
      if(d == 0) {
        count++;
      }
      if(d <= 0)
        a = chunk->dev_neighbor_i[++i_it]; 
      if(d >= 0)
        b = chunk->dev_neighbor_j[++j_it];
    }
  }
  results[blockDim.x * blockIdx.x + threadIdx.x] = count;
}

__global__ void CalculateTriangles_v2(TrCountingGraphChunk* chunk, int n, uint64_t* results, int deviceCount = 1,
    int deviceIdx = 0) {
    int from =
    gridDim.x * blockDim.x * deviceIdx +
    blockDim.x * blockIdx.x +
    threadIdx.x;
  int step = deviceCount * gridDim.x * blockDim.x;
  
  uint64_t count = 0;
  for (int k = from; k < n; k += step) {
    int i =  chunk->dev_neighbor_start[k]; 
    int j =  chunk->dev_neighbor[k]; 
    if (j==2147483647) continue;
    int64_t j_it = chunk->dev_offset[j];
    int64_t i_it = chunk->dev_offset[i];
    int64_t j_it_end = j_it+chunk->dev_degree[j]-1;
    int64_t i_it_end = i_it+chunk->dev_degree[i]-1;

    int a = chunk->dev_neighbor[i_it], b = chunk->dev_neighbor[j_it]; 
    while(j_it <= j_it_end && i_it <= i_it_end){
      int d = a-b;
      if ( d == 0 ){
        count++;
      }
      if (d <= 0)
        a = chunk->dev_neighbor[++i_it]; 
      if (d >= 0)
        b = chunk->dev_neighbor[++j_it];
    }
  }
  results[blockDim.x * blockIdx.x + threadIdx.x] = count;
}

__global__ void CalculateTriangles(
    int m, const int* __restrict__ edges, const int* __restrict__ nodes,
    uint64_t* results, int deviceCount = 1, int deviceIdx = 0) {
  int from =
    gridDim.x * blockDim.x * deviceIdx +
    blockDim.x * blockIdx.x +
    threadIdx.x;
  int step = deviceCount * gridDim.x * blockDim.x;
  uint64_t count = 0;

  for (int i = from; i < m; i += step) {
    int u = edges[i], v = edges[m + i];

    int u_it = nodes[u], u_end = nodes[u + 1];
    int v_it = nodes[v], v_end = nodes[v + 1];

    int a = edges[u_it], b = edges[v_it];
    while (u_it < u_end && v_it < v_end) {
      int d = a - b;
      if (d <= 0)
        a = edges[++u_it];
      if (d >= 0)
        b = edges[++v_it];
      if (d == 0)
        ++count;
    }
  }

  results[blockDim.x * blockIdx.x + threadIdx.x] = count;
}

void CudaAssert(hipError_t status, const char* code, const char* file, int l) {
  if (status == hipSuccess) return;
  cerr << "Cuda error: " << code << ", file " << file << ", line " << l << endl;
  exit(1);
}

#define CUCHECK(x) CudaAssert(x, #x, __FILE__, __LINE__)

int NumberOfMPs() {
  int dev, val;
  CUCHECK(hipGetDevice(&dev));
  CUCHECK(hipDeviceGetAttribute(&val, hipDeviceAttributeMultiprocessorCount, dev));
  return val;
}

size_t GlobalMemory() {
  int dev;
  hipDeviceProp_t prop;
  CUCHECK(hipGetDevice(&dev));
  CUCHECK(hipGetDeviceProperties(&prop, dev));
  return prop.totalGlobalMem;
}



uint64_t MultiGPUCalculateTriangles(
    int n, int m, int* dev_edges, int* dev_nodes, int device_count) {
  vector<int*> multi_dev_edges(device_count);
  vector<int*> multi_dev_nodes(device_count);

  multi_dev_edges[0] = dev_edges;
  multi_dev_nodes[0] = dev_nodes;

  for (int i = 1; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipMalloc(&multi_dev_edges[i], m * 2 * sizeof(int)));
    CUCHECK(hipMalloc(&multi_dev_nodes[i], (n + 1) * sizeof(int)));
    int dst = i, src = (i + 1) >> 2;
    CUCHECK(hipMemcpyPeer(
          multi_dev_edges[dst], dst, multi_dev_edges[src], src,
          m * 2 * sizeof(int)));
    CUCHECK(hipMemcpyPeer(
          multi_dev_nodes[dst], dst, multi_dev_nodes[src], src,
          (n + 1) * sizeof(int)));
  }

  vector<int> NUM_BLOCKS(device_count);
  vector<uint64_t*> multi_dev_results(device_count);

  for (int i = 0; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    NUM_BLOCKS[i] = NUM_BLOCKS_PER_MP * NumberOfMPs();
    CUCHECK(hipMalloc(
          &multi_dev_results[i],
          NUM_BLOCKS[i] * NUM_THREADS * sizeof(uint64_t)));
  }

  for (int i = 0; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(CalculateTriangles), hipFuncCachePreferL1));
    CalculateTriangles<<<NUM_BLOCKS[i], NUM_THREADS>>>(
        m, multi_dev_edges[i], multi_dev_nodes[i], multi_dev_results[i],
        device_count, i);
  }

  uint64_t result = 0;

  for (int i = 0; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipDeviceSynchronize());
    result += SumResults(NUM_BLOCKS[i] * NUM_THREADS, multi_dev_results[i]);
  }

  for (int i = 1; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipFree(multi_dev_edges[i]));
    CUCHECK(hipFree(multi_dev_nodes[i]));
  }

  for (int i = 0; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipFree(multi_dev_results[i]));
  }

  hipSetDevice(0);
  return result;
}

uint64_t GpuForward(int* edges, int num_nodes, uint64_t num_edges) {
  return MultiGpuForward(edges, 1, num_nodes, num_edges);
}

uint64_t GpuForward_v2(const TrCountingGraph& TrCountingGraph) {
    const int NUM_BLOCKS = NUM_BLOCKS_PER_MP * NumberOfMPs();
    TrCountingGraphChunk chunk(TrCountingGraph, 1);

    uint64_t* dev_results;
    CUCHECK(hipMalloc(&dev_results,
          NUM_BLOCKS * NUM_THREADS * sizeof(uint64_t)));
    
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(CalculateTriangles_v2), hipFuncCachePreferL1);
    CalculateTriangles_v2<<<NUM_BLOCKS, NUM_THREADS>>>(chunk.dev_this, 
        TrCountingGraph.offset[TrCountingGraph.nodeid_max+1], dev_results);
    CUCHECK(hipDeviceSynchronize());
    uint64_t result = SumResults(NUM_BLOCKS * NUM_THREADS, dev_results);
    return result;
}

int GetDevNum(){
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  return deviceCount;
}

int GetSplitNum(int num_nodes, uint64_t num_edges) {
  int DevNum = GetDevNum();
  int split_num = 0;
  int split_num_new = 0;
  for (int i = 0; i < DevNum; i++){
    CUCHECK(hipSetDevice(i));
    uint64_t mem = (uint64_t)GlobalMemory();  // in Byte
    mem -= (uint64_t)num_nodes * 16;  // uint64_t
    split_num_new = (int)(1 + 12 * (num_edges) / mem);
    split_num = split_num > split_num_new ? split_num : split_num_new;
  }
  return split_num;
}

void InitializeGPUMemory() {
    
}

uint64_t GpuForwardSplit_v2(const TrCountingGraph& TrCountingGraph, 
    int split_num, int64_t cpu_offset,
    int gpu_offset_start, int gpu_offset_end, int rank, int step) {
    CUCHECK(hipSetDevice(0));
    const int NUM_BLOCKS = NUM_BLOCKS_PER_MP * NumberOfMPs();
    
    TrCountingGraphChunk chunk(TrCountingGraph, split_num);

    uint64_t* dev_results;
    CUCHECK(hipMalloc(&dev_results, NUM_BLOCKS * NUM_THREADS * sizeof(uint64_t)));

    int64_t result=0;
    for(int ij = gpu_offset_start; ij < gpu_offset_end; ij++) {
        int i, j;
        get_i_j(split_num, ij, &i, &j);
        if(chunk.split_offset[i] >= cpu_offset)
            break;
        chunk.initChunk(i, j);
        int length = cpu_offset>chunk.split_offset[i+1]?chunk.split_offset[i+1]-chunk.split_offset[i]:cpu_offset-chunk.split_offset[i];
        CalculateTrianglesSplit_v2<<<NUM_BLOCKS, NUM_THREADS>>>(chunk.dev_this, length, dev_results);
        CUCHECK(hipDeviceSynchronize());
        result = result + SumResults(NUM_BLOCKS * NUM_THREADS, dev_results);
    }
    return result;
}

uint64_t MultiGpuForward(int* edges, int device_count, int num_nodes, uint64_t num_edges) {
  CUCHECK(hipSetDevice(0));
  const int NUM_BLOCKS = NUM_BLOCKS_PER_MP * NumberOfMPs();

  uint64_t m = num_edges;
  int n = num_nodes;

  int* dev_edges;
  int* dev_nodes;

  
  int* dev_temp;
  CUCHECK(hipMalloc(&dev_temp, m * 2 * sizeof(int)));
  CUCHECK(hipMemcpy(
      dev_temp, edges, m * 2 * sizeof(int), hipMemcpyHostToDevice));
  CUCHECK(hipDeviceSynchronize());
  // Memcpy edges from host to device
  SortEdges(m, dev_temp);
  CUCHECK(hipDeviceSynchronize());
  // Sort edges

  CUCHECK(hipMalloc(&dev_edges, m * 2 * sizeof(int)));
  UnzipEdges<<<NUM_BLOCKS, NUM_THREADS>>>(m, dev_temp, dev_edges);
  CUCHECK(hipFree(dev_temp));
  CUCHECK(hipDeviceSynchronize());
  // Unzip edges


  CUCHECK(hipMalloc(&dev_nodes, (n + 1) * sizeof(int)));
  CalculateNodePointers<false><<<NUM_BLOCKS, NUM_THREADS>>>(
      n, m, dev_edges, dev_nodes);
  CUCHECK(hipDeviceSynchronize());
  // Calculate nodes array for one-way unzipped edges
  uint64_t result = 0;

  if (device_count == 1) {
    uint64_t* dev_results;
    CUCHECK(hipMalloc(&dev_results,
          NUM_BLOCKS * NUM_THREADS * sizeof(uint64_t)));
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(CalculateTriangles), hipFuncCachePreferL1);
    hipProfilerStart();
    CalculateTriangles<<<NUM_BLOCKS, NUM_THREADS>>>(
        m, dev_edges, dev_nodes, dev_results);
    CUCHECK(hipDeviceSynchronize());
    hipProfilerStop();
    // Reduce
    result = SumResults(NUM_BLOCKS * NUM_THREADS, dev_results);
    CUCHECK(hipFree(dev_results));
  } else {
    result = MultiGPUCalculateTriangles(
        n, m, dev_edges, dev_nodes, device_count);
  }

  CUCHECK(hipFree(dev_edges));
  CUCHECK(hipFree(dev_nodes));
  return result;
}

void PreInitGpuContext(int device) {
  CUCHECK(hipSetDevice(device));
  CUCHECK(hipFree(NULL));
}

TrCountingGraphChunk::TrCountingGraphChunk(const TrCountingGraph& g, int split_num){
  chunk_length_max = get_split_v2(g.offset, g.nodeid_max, split_num, split_offset);
  // device memory initialization
  CUCHECK(hipMalloc(&dev_offset, (g.nodeid_max + 2) * sizeof(int64_t)));
  CUCHECK(hipMemcpy(
      dev_offset, g.offset, (g.nodeid_max + 2) * sizeof(int64_t), hipMemcpyHostToDevice));
  CUCHECK(hipMalloc(&dev_degree, (g.nodeid_max + 1) * sizeof(int)));
  CUCHECK(hipMemcpy(
      dev_degree, g.degree, (g.nodeid_max + 1) * sizeof(int), hipMemcpyHostToDevice));
  CUCHECK(hipMalloc(&dev_split_offset, (split_num + 1) * sizeof(int64_t)));
  CUCHECK(hipMemcpy(
      dev_split_offset, split_offset, (split_num + 1) * sizeof(int64_t), hipMemcpyHostToDevice));
  if(split_num==1){
    CUCHECK(hipMalloc(&dev_neighbor, (g.edge_num) * sizeof(int)));
    CUCHECK(hipMemcpy(
       dev_neighbor, g.neighboor, (g.edge_num) * sizeof(int), hipMemcpyHostToDevice));
    CUCHECK(hipMalloc(&dev_neighbor_start, (g.edge_num) * sizeof(int)));
    CUCHECK(hipMemcpy(
       dev_neighbor_start, g.neighboor_start, (g.edge_num) * sizeof(int), hipMemcpyHostToDevice));
  }
  else{
    CUCHECK(hipMalloc(&dev_neighbor_i, chunk_length_max * sizeof(int)));
    CUCHECK(hipMalloc(&dev_neighbor_start_i, chunk_length_max * sizeof(int)));
    CUCHECK(hipMalloc(&dev_neighbor_j, chunk_length_max * sizeof(int)));
  }
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(CalculateTrianglesSplit_v2), hipFuncCachePreferL1);

  Graph = &g;
  CUCHECK(hipMalloc(&dev_this, sizeof(TrCountingGraphChunk)));
  CUCHECK(hipMemcpy(dev_this,this,sizeof(TrCountingGraphChunk),hipMemcpyHostToDevice));
}
void TrCountingGraphChunk::initChunk(int i, int j){
  chunkid_i = i;
  chunkid_j = j;
  CUCHECK(hipMemcpy(
    dev_neighbor_i, Graph->neighboor+split_offset[i], (split_offset[i+1]-split_offset[i])*sizeof(int), hipMemcpyHostToDevice));
  CUCHECK(hipMemcpy(
    dev_neighbor_start_i, Graph->neighboor_start+split_offset[i], (split_offset[i+1]-split_offset[i])*sizeof(int), hipMemcpyHostToDevice));
  CUCHECK(hipMemcpy(
    dev_neighbor_j, Graph->neighboor+split_offset[j], (split_offset[j+1]-split_offset[j])*sizeof(int), hipMemcpyHostToDevice));
  CUCHECK(hipMemcpy(dev_this,this,sizeof(TrCountingGraphChunk),hipMemcpyHostToDevice));
}